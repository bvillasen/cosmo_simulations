
#include <hip/hip_runtime.h>
//Get the CIC index from the particle position ( device function )
__device__ void Get_Indexes_CIC( double xMin, double yMin, double zMin, double dx, double dy, double dz, double pos_x, double pos_y, double pos_z, int &indx_x, int &indx_y, int &indx_z ){
  indx_x = (int) floor( ( pos_x - xMin - 0.5*dx ) / dx );
  indx_y = (int) floor( ( pos_y - yMin - 0.5*dy ) / dy );
  indx_z = (int) floor( ( pos_z - zMin - 0.5*dz ) / dz );
}

__device__ double get_weight_tsc_c( double pos, double cell_center, double delta ){
  if ( fabs( (pos - cell_center)/delta ) > 0.5 ) printf( "ERROR: Distance grater than 1/2 in center weight \n");
  return 3./4 - pow( (pos - cell_center)/delta, 2 );
}
__device__ double get_weight_tsc_lr( double pos, double cell_center, double delta ){
  if ( fabs( (pos - cell_center)/delta ) < 0.5  ) printf( "ERROR: Distance lesser than 1/2 in left-right weight \n");
  if ( fabs( (pos - cell_center)/delta ) > 1.5  ) printf( "ERROR: Distance grater than 3/2 in left-right weight \n");
  return 0.5 * pow( 3./2 - fabs(pos - cell_center)/delta, 2 );
}



//Get the CIC index from the particle position ( device function )
__device__ void Get_Indexes_TSC( double xMin, double yMin, double zMin, double dx, double dy, double dz, double pos_x, double pos_y, double pos_z, int &indx_x, int &indx_y, int &indx_z ){
  indx_x = (int) floor( ( pos_x - xMin ) / dx );
  indx_y = (int) floor( ( pos_y - yMin ) / dy );
  indx_z = (int) floor( ( pos_z - zMin ) / dz );
}


extern "C"{
  
  
//CUDA Kernel to compute the TSC density from the particles positions
__global__ void Get_Density_TSC_Kernel( int n_local, double particle_mass,  double *density_dev, 
                                        double *pos_x_dev, double *pos_y_dev, double *pos_z_dev, 
                                        double xMin, double yMin, double zMin, 
                                        double xMax, double yMax, double zMax, 
                                        double dx, double dy, double dz, 
                                        int nx, int ny, int nz, int n_ghost  ){

  int tid = blockIdx.x * blockDim.x + threadIdx.x ;
  if ( tid >= n_local) return;

  int nx_g, ny_g;
  int i, j, k;
  nx_g = nx + 2*n_ghost;
  ny_g = ny + 2*n_ghost;

  double pos_x, pos_y, pos_z, pMass;
  double cell_center_x, cell_center_y, cell_center_z;
  double wx_l, wx_c, wx_r;
  double wy_l, wy_c, wy_r;
  double wz_l, wz_c, wz_r;
  double w, w_sum, wx, wy, wz;
  double dV_inv = 1./(dx*dy*dz);

  pos_x = pos_x_dev[tid];
  pos_y = pos_y_dev[tid];
  pos_z = pos_z_dev[tid];

  pMass = particle_mass * dV_inv;
  
  int indx_x, indx_y, indx_z, indx;
  Get_Indexes_TSC( xMin, yMin, zMin, dx, dy, dz, pos_x, pos_y, pos_z, indx_x, indx_y, indx_z );

  bool in_local = true;

  if ( pos_x < xMin || pos_x >= xMax ) in_local = false;
  if ( pos_y < yMin || pos_y >= yMax ) in_local = false;
  if ( pos_z < zMin || pos_z >= zMax ) in_local = false;
  if ( ! in_local  ) {
    printf(" Density CIC Error: Particle outside local domain [%f  %f  %f]  [%f %f] [%f %f] [%f %f]\n ", pos_x, pos_y, pos_z, xMin, xMax, yMin, yMax, zMin, zMax);
    return;
  }

  cell_center_x = xMin + indx_x*dx + 0.5*dx;
  cell_center_y = yMin + indx_y*dy + 0.5*dy;
  cell_center_z = zMin + indx_z*dz + 0.5*dz;
  
  // if ( tid == 0 ) printf(" d: %f\n", pow( (pos_x - cell_center_x) /dx, 2) );
  
  wx_c = get_weight_tsc_c( pos_x, cell_center_x, dx );
  wy_c = get_weight_tsc_c( pos_y, cell_center_y, dy );
  wz_c = get_weight_tsc_c( pos_z, cell_center_z, dz );
  
  wx_l = get_weight_tsc_lr( pos_x, cell_center_x - dx, dx );
  wy_l = get_weight_tsc_lr( pos_y, cell_center_y - dy, dy );
  wz_l = get_weight_tsc_lr( pos_z, cell_center_z - dz, dz );
  
  wx_r = get_weight_tsc_lr( pos_x, cell_center_x + dx, dx );
  wy_r = get_weight_tsc_lr( pos_y, cell_center_y + dy, dy );
  wz_r = get_weight_tsc_lr( pos_z, cell_center_z + dz, dz );
  
  indx_x += n_ghost;
  indx_y += n_ghost;
  indx_z += n_ghost;
  
  w_sum = 0;
  
  for ( k=-1; k<2; k++ ){
    if ( k == -1 ) wz = wz_l;
    if ( k ==  0 ) wz = wz_c;
    if ( k ==  1 ) wz = wz_r;  
  
    for ( j=-1; j<2; j++ ){
      if ( j == -1 ) wy = wy_l;
      if ( j ==  0 ) wy = wy_c;
      if ( j ==  1 ) wy = wy_r;  
  
      for ( i=-1; i<2; i++ ){
        if ( i == -1 ) wx = wx_l;
        if ( i ==  0 ) wx = wx_c;
        if ( i ==  1 ) wx = wx_r;  
  
        w = wx * wy * wz;
        indx = (indx_x+i) + (indx_y+j)*nx_g + (indx_z+k)*nx_g*ny_g; 
        atomicAdd( &density_dev[indx],  pMass * w );
        w_sum += w;
        // if ( tid == 0 ) printf(" i: %d  j: %d  k: %d   w: %f   w_sum: %f\n", i, j, k,  w, w_sum );  
      }
    }
  }
  
  if ( fabs( w_sum - 1 ) > 1e-4 ) printf("ERROR: Weight sum less than 1.   w_sum: %d", w_sum );
  
}



  
//CUDA Kernel to compute the CIC density from the particles positions
__global__ void Get_Density_CIC_Kernel( int n_local, double particle_mass,  double *density_dev, 
                                        double *pos_x_dev, double *pos_y_dev, double *pos_z_dev, 
                                        double xMin, double yMin, double zMin, 
                                        double xMax, double yMax, double zMax, 
                                        double dx, double dy, double dz, 
                                        int nx, int ny, int nz, int n_ghost  ){

  int tid = blockIdx.x * blockDim.x + threadIdx.x ;
  if ( tid >= n_local) return;

  int nx_g, ny_g;
  nx_g = nx + 2*n_ghost;
  ny_g = ny + 2*n_ghost;

  double pos_x, pos_y, pos_z, pMass;
  double cell_center_x, cell_center_y, cell_center_z;
  double delta_x, delta_y, delta_z;
  double dV_inv = 1./(dx*dy*dz);

  pos_x = pos_x_dev[tid];
  pos_y = pos_y_dev[tid];
  pos_z = pos_z_dev[tid];

  pMass = particle_mass * dV_inv;
  
  int indx_x, indx_y, indx_z, indx;
  Get_Indexes_CIC( xMin, yMin, zMin, dx, dy, dz, pos_x, pos_y, pos_z, indx_x, indx_y, indx_z );

  bool in_local = true;

  if ( pos_x < xMin || pos_x >= xMax ) in_local = false;
  if ( pos_y < yMin || pos_y >= yMax ) in_local = false;
  if ( pos_z < zMin || pos_z >= zMax ) in_local = false;
  if ( ! in_local  ) {
    printf(" Density CIC Error: Particle outside local domain [%f  %f  %f]  [%f %f] [%f %f] [%f %f]\n ", pos_x, pos_y, pos_z, xMin, xMax, yMin, yMax, zMin, zMax);
    return;
  }

  cell_center_x = xMin + indx_x*dx + 0.5*dx;
  cell_center_y = yMin + indx_y*dy + 0.5*dy;
  cell_center_z = zMin + indx_z*dz + 0.5*dz;
  delta_x = 1 - ( pos_x - cell_center_x ) / dx;
  delta_y = 1 - ( pos_y - cell_center_y ) / dy;
  delta_z = 1 - ( pos_z - cell_center_z ) / dz;
  indx_x += n_ghost;
  indx_y += n_ghost;
  indx_z += n_ghost;


  indx = indx_x + indx_y*nx_g + indx_z*nx_g*ny_g;
  // density_dev[indx] += pMass  * delta_x * delta_y * delta_z;
  atomicAdd( &density_dev[indx],  pMass  * delta_x * delta_y * delta_z);

  indx = (indx_x+1) + indx_y*nx_g + indx_z*nx_g*ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * delta_y * delta_z;
  atomicAdd( &density_dev[indx], pMass  * (1-delta_x) * delta_y * delta_z);

  indx = indx_x + (indx_y+1)*nx_g + indx_z*nx_g*ny_g;
  // density_dev[indx] += pMass  * delta_x * (1-delta_y) * delta_z;
  atomicAdd( &density_dev[indx], pMass  * delta_x * (1-delta_y) * delta_z);
  //
  indx = indx_x + indx_y*nx_g + (indx_z+1)*nx_g*ny_g;
  // density_dev[indx] += pMass  * delta_x * delta_y * (1-delta_z);
  atomicAdd( &density_dev[indx], pMass  * delta_x * delta_y * (1-delta_z) );

  indx = (indx_x+1) + (indx_y+1)*nx_g + indx_z*nx_g*ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * (1-delta_y) * delta_z;
  atomicAdd( &density_dev[indx], pMass  * (1-delta_x) * (1-delta_y) * delta_z);

  indx = (indx_x+1) + indx_y*nx_g + (indx_z+1)*nx_g*ny_g;
  // density_dev[indx] += pMass  * (1-delta_x) * delta_y * (1-delta_z);
  atomicAdd( &density_dev[indx], pMass  * (1-delta_x) * delta_y * (1-delta_z));

  indx = indx_x + (indx_y+1)*nx_g + (indx_z+1)*nx_g*ny_g;
  // density_dev[indx] += pMass  * delta_x * (1-delta_y) * (1-delta_z);
  atomicAdd( &density_dev[indx], pMass  * delta_x * (1-delta_y) * (1-delta_z));

  indx = (indx_x+1) + (indx_y+1)*nx_g + (indx_z+1)*nx_g*ny_g;
  // density_dev[indx] += pMass * (1-delta_x) * (1-delta_y) * (1-delta_z);
  atomicAdd( &density_dev[indx], pMass * (1-delta_x) * (1-delta_y) * (1-delta_z));

}

}

